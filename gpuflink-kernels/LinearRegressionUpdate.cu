
#include <hip/hip_runtime.h>
extern "C"
__global__ void linear_regression_update(double *t0, double *t1, double *r0, double *r1, int *c, int *n){
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index < *n){
		r0[index] = t0[index] / c[index];
		r1[index] = t1[index] / c[index];
	}
  	__syncthreads();
}
