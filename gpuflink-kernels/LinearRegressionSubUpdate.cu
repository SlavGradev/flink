
#include <hip/hip_runtime.h>
extern "C"
__global__ void linear_regression_sub_update(double *t0, 
					 double *t1, 
					 double *x, 
					 double *y,
					 int *r0, 
 					 int *r1,
					 int *n){
 	int index = blockIdx.x*blockDim.x + threadIdx.x;

	if(index < *n){
		r0[index] = *t0 - 0.01 * ((*t0 + (*t1 * x[index])) - y[index]);
		r1[index] = *t1 - 0.01 * (((*t0 + (*t1 * x[index])) - y[index]) * x[index]);
	}
  	__syncthreads();
}
